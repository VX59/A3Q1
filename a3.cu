#include "hip/hip_runtime.h"
#include "a3.hpp"
#include <numeric>

__device__ float K(float x)
{
    return (1/(sqrtf(2*M_PI)))*expf(-(x*x)/2);
}

__global__ void kde_kernel(const float *x, float *y, int n, int h, float k)
{
    int bx = blockIdx.x;
    int idx = blockDim.x * bx + threadIdx.x;

    if (idx < n)
    {
        float xi = x[idx];

        for (int j = 0; j < n; j++)
        {
            int xj = x[j];
            y[idx] += K((xi - xj) / h);
        }
    }
}

void gaussian_kde(int n, float h, const std::vector<float>& x, std::vector<float>& y) {

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);

    int threadsPerBlock = int(h);

    h = 0.01;

    int xblocks = (n + threadsPerBlock - 1)/ threadsPerBlock;
   
    float *d_x, *d_y;

    hipMalloc(&d_x, sizeof(float)*n);
    hipMemcpy(d_x, x.data(), n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_y, sizeof(float)*n);
    
    float k = 1/(n*h);
    
    hipDeviceSynchronize();    

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    kde_kernel<<<xblocks, threadsPerBlock, threadsPerBlock*sizeof(float)>>>(d_x, d_y, n, h, k);

    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    hipMemcpy(y.data(), d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

} // gaussian_kde